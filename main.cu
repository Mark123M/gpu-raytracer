#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "math/vec3.h"
#include "math/ray.h"
#include "math/util.h"
#include "util/color.h"
#include "entity.h"
#include "entitylist.h"
#include "material.h"
#include "sphere.h"
#include "camera.h"

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
    entity_list world;
    
    auto material_ground = std::make_shared<lambertian>(color(0.8, 0.8, 0.0));
    auto material_center = std::make_shared<lambertian>(color(0.1, 0.2, 0.5));
    auto material_left = std::make_shared<metal>(color(0.8, 0.8, 0.8), 0.3);
    auto material_right = std::make_shared<metal>(color(0.8, 0.6, 0.2), 1.0);

    world.objects.push_back(std::make_shared<sphere>(point3(0, -100.5, -1), 100, material_ground));
    world.objects.push_back(std::make_shared<sphere>(point3(0, 0, -1.2), 0.5, material_center));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.5, material_left));
    world.objects.push_back(std::make_shared<sphere>(point3(1.0, 0, -1.0), 0.5, material_right));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 225;
    cam.pixel_samples = 100;
    cam.max_depth = 50;
    cam.render(world);

    return 0;
}