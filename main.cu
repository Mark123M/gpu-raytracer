#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "math/vec3.h"
#include "math/ray.h"
#include "math/util.h"
#include "util/color.h"
#include "entity.h"
#include "entitylist.h"
#include "material.h"
#include "sphere.h"
#include "quad.h"
#include "camera.h"
#include "bvh.h"
#include "math/transform.h"

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

void spheres() {
    entity_list world;

    /*auto material_ground = std::make_shared<lambertian>(color(0.8, 0.8, 0.0));
    auto material_center = std::make_shared<lambertian>(color(0.1, 0.2, 0.5));
    auto material_left = std::make_shared<dielectric>(1.50); // Air bubble in water //std::make_shared<metal>(color(0.8, 0.8, 0.8), 0.3);
    auto material_bubble = std::make_shared<dielectric>(1.00 / 1.50); // Material index over medium index (air in glass)
    auto material_right = std::make_shared<metal>(color(0.8, 0.6, 0.2), 1.0);

    world.objects.push_back(std::make_shared<sphere>(point3(0, -100.5, -1), 100, material_ground));
    world.objects.push_back(std::make_shared<sphere>(point3(0, 0, -1.2), 0.5, material_center));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.5, material_left));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.4, material_bubble));
    world.objects.push_back(std::make_shared<sphere>(point3(1.0, 0, -1.0), 0.5, material_right)); */

    auto ground_material = std::make_shared<lambertian>(color(0.5, 0.5, 0.5));
    world.add(std::make_shared<sphere>(point3(0, -1000, 0), 1000, ground_material));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            float choose_mat = randf();
            point3 center(a + 0.9 * randf(), 0.2, b + 0.9 * randf());

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                std::shared_ptr<material> sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color{ randf(), randf(), randf() } *color{ randf(), randf(), randf() };
                    sphere_material = std::make_shared<lambertian>(albedo);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color{ randf(0.5, 1), randf(0.5, 1), randf(0.5, 1) };
                    auto fuzz = randf(0, 0.5);
                    sphere_material = std::make_shared<metal>(albedo, fuzz);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = std::make_shared<dielectric>(1.5);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = std::make_shared<dielectric>(1.5);
    world.add(std::make_shared<sphere>(point3(0, 1, 0), 1.0, material1));

    auto material2 = std::make_shared<lambertian>(color(0.4, 0.2, 0.1));
    world.add(std::make_shared<sphere>(point3(-4, 1, 0), 1.0, material2));

    auto material3 = std::make_shared<metal>(color(0.7, 0.6, 0.5), 0.0);
    world.add(std::make_shared<sphere>(point3(4, 1, 0), 1.0, material1));

    world = entity_list(std::make_shared<bvh>(world));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 675;
    cam.pixel_samples = 100;
    cam.max_depth = 20;

    cam.vfov = 20;
    cam.origin = point3(13, 2, 3);
    cam.lookat = point3(0, 0, 0);
    cam.vup = vec3(0, 1, 0);

    std::vector<entity*> lights = {};
    cam.render(world, lights);
}

void quads() {
    entity_list world;

    // Materials
    auto left_red = std::make_shared<lambertian>(color(1.0, 0.2, 0.2));
    auto back_green = std::make_shared<lambertian>(color(0.2, 1.0, 0.2));
    auto right_blue = std::make_shared<lambertian>(color(0.2, 0.2, 1.0));
    auto upper_orange = std::make_shared<lambertian>(color(1.0, 0.5, 0.0));
    auto lower_teal = std::make_shared<lambertian>(color(0.2, 0.8, 0.8));

    // Quads
    world.add(std::make_shared<quad>(point3(-3, -2, 5), vec3(0, 0, -4), vec3(0, 4, 0), left_red, nullptr));
    world.add(std::make_shared<quad>(point3(-2, -2, 0), vec3(4, 0, 0), vec3(0, 4, 0), back_green, nullptr));
    world.add(std::make_shared<quad>(point3(3, -2, 1), vec3(0, 0, 4), vec3(0, 4, 0), right_blue, nullptr));
    world.add(std::make_shared<quad>(point3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), upper_orange, nullptr));
    world.add(std::make_shared<quad>(point3(-2, -3, 5), vec3(4, 0, 0), vec3(0, 0, -4), lower_teal,nullptr));

    camera cam;
    cam.aspect_ratio = 1.0;
    cam.image_height = 400;
    cam.pixel_samples = 100;
    cam.max_depth = 50;

    cam.vfov = 80;
    cam.origin = point3(0, 0, 9);
    cam.lookat = point3(0, 0, 0);
    cam.vup = vec3(0, 1, 0);

    std::vector<entity*> lights = {};
    cam.render(world, lights);
}

void cornell_box() {
    entity_list world;

    auto red = std::make_shared<lambertian>(color(.65, .05, .05));
    auto white = std::make_shared<lambertian>(color(.73, .73, .73));
    auto green = std::make_shared<lambertian>(color(.12, .45, .15));
    auto light = std::make_shared<area_light> (color(15, 15, 15), world); // I can't share lights, because lights take a shape which is set in the quad constructor....will be overwritten

    world.add(std::make_shared<quad>(point3(555, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), green, nullptr));
    world.add(std::make_shared<quad>(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), red, nullptr));
    world.add(std::make_shared<quad>(point3(0, 0, 0), vec3(555, 0, 0), vec3(0, 0, 555), white, nullptr));
    world.add(std::make_shared<quad>(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), white, nullptr));
    world.add(std::make_shared<quad>(point3(0, 0, 555), vec3(555, 0, 0), vec3(0, 555, 0), white, nullptr));

    auto light1 = std::make_shared<quad>(point3(343, 554, 332), vec3(-130, 0, 0), vec3(0, 0, -105), nullptr, light);
    world.add(light1);

    std::shared_ptr<entity> box1 = box(point3(0, 0, 0), point3(165, 330, 165), white);
    box1 = std:: make_shared<rotate_y>(box1, 15);
    box1 = std::make_shared<translate>(box1, vec3(265, 0, 295));
    world.add(box1);

    std::shared_ptr<entity> box2 = box(point3(0, 0, 0), point3(165, 165, 165), white);
    box2 = std::make_shared<rotate_y>(box2, -18);
    box2 = std::make_shared<translate>(box2, vec3(130, 0, 65));
    world.add(box2);

    world = entity_list(std::make_shared<bvh>(world));

    camera cam;
    cam.aspect_ratio = 1.0;
    cam.image_height = 600;
    cam.pixel_samples = 10;
    cam.max_depth = 16;

    cam.vfov = 40;
    cam.origin = point3(278, 278, -800);
    cam.lookat = point3(278, 278, 0);
    cam.vup = vec3(0, 1, 0);

    std::vector<entity*> lights = {light1.get()};
    cam.render(world, lights);
}

void light_test() {
    entity_list world;

    auto light_mat = std::make_shared<area_light>(color{15.0, 15.0, 15.0}, world);
    auto sphere_mat = std::make_shared<lambertian>(color{ 0.96078, 0.67059, 0.72549 });
    auto ground_mat = std::make_shared<lambertian>(color{ 0.35686, 0.81176, 0.98431 });
    world.add(std::make_shared<sphere>(point3(0, -1000, 0), 1000, ground_mat));
    //world.add(std::make_shared<quad>(point3(-250, 0, -250), vec3(555, 0, 0), vec3(0, 0, 555), ground_mat, nullptr));
    world.add(std::make_shared<sphere>(point3(0, 2, 0), 2, sphere_mat));

    auto light1 = std::make_shared<quad>(point3(3, 1, -2), vec3(2, 0, 0), vec3(0, 2, 0), nullptr, light_mat);
    world.add(light1);

    //world = entity_list(std::make_shared<bvh>(world));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 400;
    cam.pixel_samples = 100;
    cam.max_depth = 16;

    cam.vfov = 20;
    cam.origin = point3(26, 3, 6);
    cam.lookat = point3(0, 2, 0);
    cam.vup = vec3(0, 1, 0);

    std::vector<entity*> lights = { light1.get() };
    cam.render(world, lights);
}

void light_test_2() {
    entity_list world;

    auto white = std::make_shared<lambertian>(color(.73, .73, .73));
    auto light = std::make_shared<area_light>(color(15, 15, 15), world);

    auto light1 = std::make_shared<quad>(point3(143, 254, 332), vec3(0, 130, 0), vec3(0, 0, 105), nullptr, light);
    world.add(light1);
    world.add(std::make_shared<quad>(point3(0, 0, 0), vec3(555, 0, 0), vec3(0, 0, 555), white, nullptr));

    camera cam;
    cam.aspect_ratio = 1.0;
    cam.image_height = 600;
    cam.pixel_samples = 100;
    cam.max_depth = 20;

    cam.vfov = 40;
    cam.origin = point3(278, 278, -800);
    cam.lookat = point3(278, 278, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world, {light1.get()});
}

void math_test() {
    vec3 v = { 5.4, -4.05, 3.3333 };

    transform id{ vec3{1, 0, 0}, vec3{0, 1, 0}, vec3{0, 0, 1}, vec3{0, 0, 0} };
    std::cout << id.local_to_world(v) << std::endl;
    std::cout << id.world_to_local(v) << std::endl;
    std::cout << id << std::endl;

    transform t1{ vec3{0, 0, -1}, normalize(vec3{1, 1, 0}), normalize(vec3{1, -1, 0}), vec3{-4, 2, 1} };
    std::cout << t1.local_to_world(v) << std::endl;
    std::cout << t1.world_to_local(t1.local_to_world(v)) << std::endl;
    std::cout << t1 << std::endl;
}

int main() {
    //math_test();
    //spheres();
    //quads();
    //light_test();
    //light_test_2();
    cornell_box();
    return 0;
}