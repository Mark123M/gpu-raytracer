#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "math/vec3.h"
#include "math/ray.h"
#include "math/util.h"
#include "util/color.h"
#include "entity.h"
#include "entitylist.h"
#include "material.h"
#include "sphere.h"
#include "quad.h"
#include "camera.h"
#include "bvh.h"

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

void spheres() {
    entity_list world;

    /*auto material_ground = std::make_shared<lambertian>(color(0.8, 0.8, 0.0));
    auto material_center = std::make_shared<lambertian>(color(0.1, 0.2, 0.5));
    auto material_left = std::make_shared<dielectric>(1.50); // Air bubble in water //std::make_shared<metal>(color(0.8, 0.8, 0.8), 0.3);
    auto material_bubble = std::make_shared<dielectric>(1.00 / 1.50); // Material index over medium index (air in glass)
    auto material_right = std::make_shared<metal>(color(0.8, 0.6, 0.2), 1.0);

    world.objects.push_back(std::make_shared<sphere>(point3(0, -100.5, -1), 100, material_ground));
    world.objects.push_back(std::make_shared<sphere>(point3(0, 0, -1.2), 0.5, material_center));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.5, material_left));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.4, material_bubble));
    world.objects.push_back(std::make_shared<sphere>(point3(1.0, 0, -1.0), 0.5, material_right)); */

    auto ground_material = std::make_shared<lambertian>(color(0.5, 0.5, 0.5));
    world.add(std::make_shared<sphere>(point3(0, -1000, 0), 1000, ground_material));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            float choose_mat = randf();
            point3 center(a + 0.9 * randf(), 0.2, b + 0.9 * randf());

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                std::shared_ptr<material> sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color{ randf(), randf(), randf() } *color{ randf(), randf(), randf() };
                    sphere_material = std::make_shared<lambertian>(albedo);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color{ randf(0.5, 1), randf(0.5, 1), randf(0.5, 1) };
                    auto fuzz = randf(0, 0.5);
                    sphere_material = std::make_shared<metal>(albedo, fuzz);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = std::make_shared<dielectric>(1.5);
                    world.add(std::make_shared<sphere>(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = std::make_shared<dielectric>(1.5);
    world.add(std::make_shared<sphere>(point3(0, 1, 0), 1.0, material1));

    auto material2 = std::make_shared<lambertian>(color(0.4, 0.2, 0.1));
    world.add(std::make_shared<sphere>(point3(-4, 1, 0), 1.0, material2));

    auto material3 = std::make_shared<metal>(color(0.7, 0.6, 0.5), 0.0);
    world.add(std::make_shared<sphere>(point3(4, 1, 0), 1.0, material1));

    world = entity_list(std::make_shared<bvh>(world));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 675;
    cam.pixel_samples = 100;
    cam.max_depth = 20;

    cam.vfov = 20;
    cam.origin = point3(13, 2, 3);
    cam.lookat = point3(0, 0, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world);
}

void quads() {
    entity_list world;

    // Materials
    auto left_red = std::make_shared<lambertian>(color(1.0, 0.2, 0.2));
    auto back_green = std::make_shared<lambertian>(color(0.2, 1.0, 0.2));
    auto right_blue = std::make_shared<lambertian>(color(0.2, 0.2, 1.0));
    auto upper_orange = std::make_shared<lambertian>(color(1.0, 0.5, 0.0));
    auto lower_teal = std::make_shared<lambertian>(color(0.2, 0.8, 0.8));

    // Quads
    world.add(std::make_shared<quad>(point3(-3, -2, 5), vec3(0, 0, -4), vec3(0, 4, 0), left_red));
    world.add(std::make_shared<quad>(point3(-2, -2, 0), vec3(4, 0, 0), vec3(0, 4, 0), back_green));
    world.add(std::make_shared<quad>(point3(3, -2, 1), vec3(0, 0, 4), vec3(0, 4, 0), right_blue));
    world.add(std::make_shared<quad>(point3(-2, 3, 1), vec3(4, 0, 0), vec3(0, 0, 4), upper_orange));
    world.add(std::make_shared<quad>(point3(-2, -3, 5), vec3(4, 0, 0), vec3(0, 0, -4), lower_teal));

    camera cam;

    cam.aspect_ratio = 1.0;
    cam.image_height = 400;
    cam.pixel_samples = 100;
    cam.max_depth = 50;

    cam.vfov = 80;
    cam.origin = point3(0, 0, 9);
    cam.lookat = point3(0, 0, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world);
    
}

void cornell_box() {
    entity_list world;

    auto red = std::make_shared<lambertian>(color(.65, .05, .05));
    auto white = std::make_shared<lambertian>(color(.73, .73, .73));
    auto green = std::make_shared<lambertian>(color(.12, .45, .15));
    auto light = std::make_shared<area_light> (color(15, 15, 15));

    world.add(std::make_shared<quad>(point3(555, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), green));
    world.add(std::make_shared<quad>(point3(0, 0, 0), vec3(0, 555, 0), vec3(0, 0, 555), red));
    world.add(std::make_shared<quad>(point3(343, 554, 332), vec3(-130, 0, 0), vec3(0, 0, -105), light));
    world.add(std::make_shared<quad>(point3(0, 0, 0), vec3(555, 0, 0), vec3(0, 0, 555), white));
    world.add(std::make_shared<quad>(point3(555, 555, 555), vec3(-555, 0, 0), vec3(0, 0, -555), white));
    world.add(std::make_shared<quad>(point3(0, 0, 555), vec3(555, 0, 0), vec3(0, 555, 0), white));

    std::shared_ptr<entity> box1 = box(point3(0, 0, 0), point3(165, 330, 165), white);
    box1 = std:: make_shared<rotate_y>(box1, 15);
    box1 = std::make_shared<translate>(box1, vec3(265, 0, 295));
    world.add(box1);

    std::shared_ptr<entity> box2 = box(point3(0, 0, 0), point3(165, 165, 165), white);
    box2 = std::make_shared<rotate_y>(box2, -18);
    box2 = std::make_shared<translate>(box2, vec3(130, 0, 65));
    world.add(box2);

    camera cam;

    cam.aspect_ratio = 1.0;
    cam.image_height = 600;
    cam.pixel_samples = 200;
    cam.max_depth = 50;

    cam.vfov = 40;
    cam.origin = point3(278, 278, -800);
    cam.lookat = point3(278, 278, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world);
}

void light() {
    entity_list world;

    auto light_mat = std::make_shared<area_light>(color{5.0, 5.0, 5.0});
    auto sphere_mat = std::make_shared<lambertian>(color{ 0.96078, 0.67059, 0.72549 });
    auto ground_mat = std::make_shared<lambertian>(color{ 0.35686, 0.81176, 0.98431 });
    world.add(std::make_shared<sphere>(point3(0, -1000, 0), 1000, ground_mat));
    world.add(std::make_shared<sphere>(point3(0, 2, 0), 2, sphere_mat));
    world.add(std::make_shared<quad>(point3(3, 1, -2), vec3(2, 0, 0), vec3(0, 2, 0), light_mat));

    camera cam;

    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 400;
    cam.pixel_samples = 100;
    cam.max_depth = 50;

    cam.vfov = 20;
    cam.origin = point3(26, 3, 6);
    cam.lookat = point3(0, 2, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world);
}

int main() {
    //spheres();
    //quads();
    //light();
    cornell_box();
    return 0;
}