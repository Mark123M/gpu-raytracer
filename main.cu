#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "math/vec3.h"
#include "math/ray.h"
#include "math/util.h"
#include "util/color.h"
#include "entity.h"
#include "entitylist.h"
#include "sphere.h"
#include "camera.h"

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
    entity_list world;
    world.objects.push_back(std::make_shared<sphere>(point3(0, 0, -1), 0.5));
    world.objects.push_back(std::make_shared<sphere>(point3(0, -100.5, -1), 100));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 225;
    cam.pixel_samples = 100;
    cam.max_depth = 50;
    cam.render(world);

    return 0;
}