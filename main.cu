#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>

#include "hip/hip_runtime.h"
#include ""
#include "math/vec3.h"
#include "math/ray.h"
#include "math/util.h"
#include "util/color.h"
#include "entity.h"
#include "entitylist.h"
#include "material.h"
#include "sphere.h"
#include "camera.h"

__global__ void hello() {
    printf("Hello from block: %u, thread: %u\n", blockIdx.x, threadIdx.x);
}

int main() {
    entity_list world;
    
    /*auto material_ground = std::make_shared<lambertian>(color(0.8, 0.8, 0.0));
    auto material_center = std::make_shared<lambertian>(color(0.1, 0.2, 0.5));
    auto material_left = std::make_shared<dielectric>(1.50); // Air bubble in water //std::make_shared<metal>(color(0.8, 0.8, 0.8), 0.3);
    auto material_bubble = std::make_shared<dielectric>(1.00 / 1.50); // Material index over medium index (air in glass)
    auto material_right = std::make_shared<metal>(color(0.8, 0.6, 0.2), 1.0);

    world.objects.push_back(std::make_shared<sphere>(point3(0, -100.5, -1), 100, material_ground));
    world.objects.push_back(std::make_shared<sphere>(point3(0, 0, -1.2), 0.5, material_center));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.5, material_left));
    world.objects.push_back(std::make_shared<sphere>(point3(-1.0, 0, -1.0), 0.4, material_bubble));
    world.objects.push_back(std::make_shared<sphere>(point3(1.0, 0, -1.0), 0.5, material_right)); */

    auto ground_material = std::make_shared<lambertian>(color(0.5, 0.5, 0.5));
    world.objects.push_back(std::make_shared<sphere>(point3(0, -1000, 0), 1000, ground_material));

    for (int a = -11; a < 11; a++) {
        for (int b = -11; b < 11; b++) {
            float choose_mat = randf();
            point3 center(a + 0.9 * randf(), 0.2, b + 0.9 * randf());

            if ((center - point3(4, 0.2, 0)).length() > 0.9) {
                std::shared_ptr<material> sphere_material;

                if (choose_mat < 0.8) {
                    // diffuse
                    auto albedo = color{ randf(), randf(), randf() } * color{randf(), randf(), randf()};
                    sphere_material = std::make_shared<lambertian>(albedo);
                    world.objects.push_back(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else if (choose_mat < 0.95) {
                    // metal
                    auto albedo = color{randf(0.5, 1), randf(0.5, 1), randf(0.5, 1)};
                    auto fuzz = randf(0, 0.5);
                    sphere_material = std::make_shared<metal>(albedo, fuzz);
                    world.objects.push_back(std::make_shared<sphere>(center, 0.2, sphere_material));
                } else {
                    // glass
                    sphere_material = std::make_shared<dielectric>(1.5);
                    world.objects.push_back(std::make_shared<sphere>(center, 0.2, sphere_material));
                }
            }
        }
    }

    auto material1 = std::make_shared<dielectric>(1.5);
    world.objects.push_back(std::make_shared<sphere>(point3(0, 1, 0), 1.0, material1));

    auto material2 = std::make_shared<lambertian>(color(0.4, 0.2, 0.1));
    world.objects.push_back(std::make_shared<sphere>(point3(-4, 1, 0), 1.0, material2));

    auto material3 = std::make_shared<metal>(color(0.7, 0.6, 0.5), 0.0);
    world.objects.push_back(std::make_shared<sphere>(point3(4, 1, 0), 1.0, material3));

    camera cam;
    cam.aspect_ratio = 16.0 / 9.0;
    cam.image_height = 675;
    cam.pixel_samples = 100;
    cam.max_depth = 20;

    cam.vfov = 20;
    cam.origin = point3(13, 2, 3);
    cam.lookat = point3(0, 0, 0);
    cam.vup = vec3(0, 1, 0);

    cam.render(world);
    return 0;
}